#include "hip/hip_runtime.h"
/*****************************************************
 * CG Solver (HPC Software Lab)
 *
 * Parallel Programming Models for Applications in the 
 * Area of High-Performance Computation
 *====================================================
 * IT Center (ITC)
 * RWTH Aachen University, Germany
 * Author: Tim Cramer (cramer@itc.rwth-aachen.de)
 * 	   Fabian Schneider (f.schneider@itc.rwth-aachen.de)
 * Date: 2010 - 2015
 *****************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#ifdef _OPENACC
# include <openacc.h>
#endif

#ifdef CUDA
# include <hip/hip_runtime.h>
#endif

#include "solver.h"
#include "output.h"
#include "reduction.h"

#ifndef NO_ERROR_CHECKS
#define CHECK_CUDA_ERROR(expr) { if ((expr) != hipSuccess) { printf("Error when executing cuda function"); } } 
#else
#define CHECK_CUDA_ERROR(expr) { expr; }
#endif

void printError() {
#ifndef NO_ERROR_CHECKS
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));
	}
#endif
}

#define UNROLLED_ADD_SYNC(n) { \
if (blockSize >= n) { \
	if (tid < n/2) { \
		sdata[tid] = localSum = localSum + sdata[tid + n/2]; \
		} \
	__syncthreads(); \
	} \
}

#define UNROLLED_ADD(n) { \
	if (blockSize >= n) { \
		sdata[tid] = localSum = localSum + sdata[tid + n/2]; \
	} __syncthreads(); \
}


 template<unsigned int blockSize>
__global__ void devVectorDot(const floatType* __restrict__ const a, const floatType* __restrict__ const b, const int n, floatType* __restrict__ const ab){
	extern __shared__ floatType sdata[];
	const int tid = threadIdx.x;
	int i = blockIdx.x * (blockSize*2) + tid;
	const int gridSize = blockSize*2 * gridDim.x;

	floatType localSum = 0;
	#pragma unroll 2
	while (i < n) {
		localSum += a[i]*b[i];
		#ifdef DEBUG
		printf("vecdot: %i %f %f %f\n", i, a[i], b[i], localSum);
		#endif
		if (i + blockSize < n)
	            localSum += a[i+blockSize]*b[i+blockSize];
	    	#ifdef DEBUG
    		printf("vecdot: %f\n",  localSum);
    		#endif
		i += gridSize;
	}
	sdata[tid] = localSum;
	__syncthreads();

	UNROLLED_ADD_SYNC(1024)
	UNROLLED_ADD_SYNC(512)
	UNROLLED_ADD_SYNC(256)
	UNROLLED_ADD_SYNC(128)	

	if (tid < 32) {
		UNROLLED_ADD(64)
		UNROLLED_ADD(32)
		UNROLLED_ADD(16)
		UNROLLED_ADD(8)
		UNROLLED_ADD(4)
		UNROLLED_ADD(2)
	}

	
	if (tid == 0) {
		ab[blockIdx.x] = localSum;
	}

}

/* a <- <x,x> */
template<unsigned int blockSize>
__global__ void devVectorSquare(const floatType* __restrict__ const x, const int n, floatType* __restrict__ const a){
	extern __shared__ floatType sdata[];
	const int tid = threadIdx.x;
	int i = blockIdx.x * (blockSize*2) + tid;
	const int gridSize = blockSize*2 * gridDim.x;

	floatType localSum = 0;

	#pragma unroll 2
	while (i < n) {
		localSum += x[i]*x[i];
		if (i + blockSize < n)
	            localSum += x[i+blockSize]*x[i+blockSize];
		i += gridSize;
	}
	sdata[tid] = localSum;
	__syncthreads();

	UNROLLED_ADD_SYNC(1024)
	UNROLLED_ADD_SYNC(512)
	UNROLLED_ADD_SYNC(256)
	UNROLLED_ADD_SYNC(128)

	if (tid < 32) {
		UNROLLED_ADD(64)
		UNROLLED_ADD(32)
		UNROLLED_ADD(16)
		UNROLLED_ADD(8)
		UNROLLED_ADD(4)
		UNROLLED_ADD(2)
	}

	if (tid == 0) {
		a[blockIdx.x] = localSum;
	}
}


/* y <- ax + y */
__global__ void axpy(const floatType a, const floatType* __restrict__ const x, const int n, floatType* __restrict__ const y){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i]=a*x[i]+y[i];
	}
}

/* y <- x + ay */
__global__ void xpay(const floatType* __restrict__ const x, const floatType a, const int n, floatType* __restrict__ const y){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i]=x[i]+a*y[i];
	}
}


/* y <- A*x
 * Remember that A is stored in the ELLPACK-R format (data, indices, length, n, nnz, maxNNZ). */
__global__ void matvec(const int n, const int nnz, const int maxNNZ, const floatType* __restrict__ const data, const int* __restrict__ const indices, const int* __restrict__ const length, const floatType* __restrict__ const x, floatType* __restrict__ const y){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col;
	if (row < n) {
		floatType temp = 0;	
		#pragma unroll 2
		for (col = 0; col < length[row]; col++) {
			int k = col * n + row;
			temp += data[k] * x[indices[k]];
		}
		y[row] = temp;
	}
}

// texref version
// define a global texref for matvec. has to be int2 for double
texture<int2, hipTextureType1D, hipReadModeElementType> devPRef;
__global__ void matvecRef(const int n, const int nnz, const int maxNNZ, const floatType* __restrict__ const data, const int* __restrict__ const indices, const int* __restrict__ const length, floatType* __restrict__ const y){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col;
	if (row < n) {
		floatType temp = 0;	
		#pragma unroll 2
		for (col = 0; col < length[row]; col++) {
			int k = col * n + row;
			int2 pv = tex1Dfetch(devPRef, indices[k]);
			floatType val = __hiloint2double(pv.y, pv.x);
			temp += data[k] * val;
		}
		y[row] = temp;
	}
}


// optimize a bit for G3
#define REDUCTION_BLOCK_SIZE 128
#define REDUCTION_BLOCK_COUNT(n) 64

void vectorSquare(const floatType* __restrict__ const x, const int n, floatType* __restrict__ a) {
	const int threadsPerBlock = REDUCTION_BLOCK_SIZE;
	const int numBlocks = REDUCTION_BLOCK_COUNT(n);
	const size_t size = numBlocks*sizeof(floatType);

	// hacky as fuck, but works
	static floatType* out = NULL;
	static floatType* devOut = NULL;

	if (devOut == NULL) {
		CHECK_CUDA_ERROR(hipHostAlloc(&out, size, hipHostMallocMapped));
		CHECK_CUDA_ERROR(hipHostGetDevicePointer(&devOut, out, 0));
	}
	
	devVectorSquare<threadsPerBlock><<<numBlocks, threadsPerBlock, threadsPerBlock*sizeof(floatType)>>>(x, n, devOut);
	printError();
	hipDeviceSynchronize();

	*a = reduce(out, numBlocks);
}

void vectorDot(const floatType* __restrict__ const a, const floatType* __restrict__ const b, const int n, floatType* __restrict__ const ab) {
	const int threadsPerBlock = REDUCTION_BLOCK_SIZE;
	const int numBlocks = REDUCTION_BLOCK_COUNT(n);
	const size_t size = numBlocks*sizeof(floatType);

	// hacky as fuck, but works
	static floatType* out = NULL;
	static floatType* devOut = NULL;

	if (devOut == NULL) {
		CHECK_CUDA_ERROR(hipHostAlloc(&out, size, hipHostMallocMapped));
		CHECK_CUDA_ERROR(hipHostGetDevicePointer(&devOut, out, 0));
	}

	devVectorDot<threadsPerBlock><<<numBlocks, threadsPerBlock, threadsPerBlock*sizeof(floatType)>>>(a, b, n, devOut);
	printError();
	hipDeviceSynchronize();

	*ab = reduce(out, numBlocks);
}

void nrm2(const floatType* __restrict__ const x, const int n, floatType* __restrict__ const nrm) {
	floatType temp;
	vectorSquare(x, n, &temp);
	*nrm = rsqrt(temp);
}

__global__ void diagMult(const floatType* __restrict__ const diag, const floatType* __restrict__ const x, const int n, floatType* __restrict__ const out) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		out[i] = x[i]/diag[i];
	}
}

__global__ void getDiag(const int n, const int nnz, const int maxNNZ, const floatType* __restrict__ const data, const int* __restrict__ const indices, const int* __restrict__ const length, floatType* __restrict__ const diag) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		int j;
		#pragma unroll 2
		for (j = 0; j < length[i]; j++) {
			int idx = j*n + i;
			int realcol = indices[idx];
			if (i == realcol) {
				diag[i] = data[idx];
			}
		}
	}
}

/***************************************
 *         Conjugate Gradient          *
 *   This function will do the CG      *
 *  algorithm without preconditioning. *
 *    For optimiziation you must not   *
 *        change the algorithm.        *
 ***************************************
 r(0)    = b - Ax(0)
 p(0)    = r(0)
 rho(0)    =  <r(0),r(0)>                
 ***************************************
 for k=0,1,2,...,n-1
   q(k)      = A * p(k)                 
   dot_pq    = <p(k),q(k)>             
   alpha     = rho(k) / dot_pq
   x(k+1)    = x(k) + alpha*p(k)      
   r(k+1)    = r(k) - alpha*q(k)     
   check convergence ||r(k+1)||_2 < eps  
	 rho(k+1)  = <r(k+1), r(k+1)>         
   beta      = rho(k+1) / rho(k)
   p(k+1)    = r(k+1) + beta*p(k)      
***************************************/
void cg(const int n, const int nnz, const int maxNNZ, const floatType* __restrict__ const data, const int* __restrict__ const indices, const int* __restrict__ const length, const floatType* __restrict__ const b, floatType* __restrict__ const x, struct SolverConfig* sc){
	floatType *devR, *devP, *devQ;
	floatType alpha, beta, rho, rho_old, dot_pq, bnrm2, check;
	int iter;
 	float timeMatvec_s;
 	float timeMatvec=0;

 	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	/* allocate memory */
	const size_t fvecSize = n * sizeof(floatType);
	const size_t ivecSize = n * sizeof(int);
	const size_t matCount = n * maxNNZ;
	const size_t fmatSize = matCount * sizeof(floatType);
	const size_t imatSize = matCount * sizeof(int);

	// varying block sizes 
	#define BLOCK_SIZE(func, bs)  \
		int func##BlockSize = bs; \
		int func##GridSize = (n + func##BlockSize - 1)/func##BlockSize; 
	
	#define LAUNCH(kernel) kernel<<<kernel##GridSize, kernel##BlockSize>>>

	BLOCK_SIZE(matvec, 128);
	BLOCK_SIZE(matvecRef, 128);
	BLOCK_SIZE(xpay, 128);
	BLOCK_SIZE(axpy, 128);
	BLOCK_SIZE(getDiag, 128);
	BLOCK_SIZE(diagMult, 128);



	CHECK_CUDA_ERROR(hipMalloc(&devP, fvecSize));


	CHECK_CUDA_ERROR(hipMalloc(&devR, fvecSize));

	CHECK_CUDA_ERROR(hipMalloc(&devQ, fvecSize));
	// cuda memory for arguments
	//hipChannelFormatDesc iChan = hipCreateChannelDesc<int>();
	
	floatType *devData, *devB, *devDiag, *devZ, *devX;
	int *devIndices, *devLength;

	CHECK_CUDA_ERROR(hipMalloc(&devB, fvecSize));
	CHECK_CUDA_ERROR(hipMemcpy(devB, b, fvecSize, hipMemcpyHostToDevice));

	CHECK_CUDA_ERROR(hipMalloc(&devData, fmatSize));
	CHECK_CUDA_ERROR(hipMemcpy(devData, data, fmatSize, hipMemcpyHostToDevice));

	CHECK_CUDA_ERROR(hipMalloc(&devIndices, imatSize));
	CHECK_CUDA_ERROR(hipMemcpy(devIndices, indices, imatSize, hipMemcpyHostToDevice));

	CHECK_CUDA_ERROR(hipMalloc(&devLength, ivecSize));
	CHECK_CUDA_ERROR(hipMemcpy(devLength, length, ivecSize, hipMemcpyHostToDevice));
	
	CHECK_CUDA_ERROR(hipMalloc(&devDiag, fvecSize));
	LAUNCH(getDiag)(n, nnz, maxNNZ, devData, devIndices, devLength, devDiag);
	printError();

	CHECK_CUDA_ERROR(hipMalloc(&devZ, fvecSize));

	CHECK_CUDA_ERROR(hipMalloc(&devX, fvecSize));
	CHECK_CUDA_ERROR(hipMemcpy(devX, x, fvecSize, hipMemcpyHostToDevice));

	// texture reference
	size_t offsetP;
	hipBindTexture(&offsetP, devPRef, devP, fvecSize);

	
	/* r(0)    = b - Ax(0) */
	hipEventRecord(start);
	LAUNCH(matvec)(n, nnz, maxNNZ, devData, devIndices, devLength, devX, devR);
	printError();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timeMatvec_s, start, stop);
	timeMatvec += timeMatvec_s/1000;

	LAUNCH(xpay)(devB, -1.0, n, devR);
	printError();

	
	LAUNCH(diagMult)(devDiag, devR, n, devZ);
	printError();
	vectorDot(devR, devZ, n, &rho);

	/* Calculate initial residuum */
	nrm2(devR, n, &bnrm2);
	printf("bnrm2: %e\n", bnrm2);

	/* p(0)    = z(0) */
	CHECK_CUDA_ERROR(hipMemcpy(devP, devZ, fvecSize, hipMemcpyDeviceToDevice));

	/* check(0)    =  <r(0),r(0)> */
	/* rho(0)    =  <r(0),z(0)> */
	vectorSquare(devR, n, &check);
	printf("rho_0=%e/%e\n", rho, check);
	for(iter = 0; iter < sc->maxIter; iter++){
		DBGMSG("=============== Iteration %d ======================\n", iter);
	
		/* q(k)      = A * p(k) */
		hipEventRecord(start);
		LAUNCH(matvecRef)(n, nnz, maxNNZ, devData, devIndices, devLength, devQ);
		printError();	
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&timeMatvec_s, start, stop);
		timeMatvec += timeMatvec_s/1000;

		/* dot_pq    = <p(k),q(k)> */
		vectorDot(devP, devQ, n, &dot_pq);

		/* alpha     = rho(k) / dot_pq */
		alpha = rho / dot_pq;

		/* x(k+1)    = x(k) + alpha*p(k) */
		LAUNCH(axpy)(alpha, devP, n, devX);
		printError();

		/* r(k+1)    = r(k) - alpha*q(k) */
		LAUNCH(axpy)(-alpha, devQ, n, devR);
		printError();


		rho_old = rho;

		/* rho(k+1)  = <r(k+1), z(k+1)> */
		LAUNCH(diagMult)(devDiag, devR, n, devZ);
		printError();

		vectorDot(devR, devZ, n, &rho);
		vectorSquare(devR, n, &check);

		/* Normalize the residual with initial one */
		sc->residual = sqrt(check) * bnrm2;
   	
		/* Check convergence ||r(k+1)||_2 < eps
		 * If the residual is smaller than the CG
		 * tolerance specified in the CG_TOLERANCE
		 * environment variable our solution vector
		 * is good enough and we can stop the 
		 * algorithm. */
		#ifdef DEBUG
		#define RESIDUAL_DEBUG
		#endif
		#ifdef RESIDUAL_DEBUG
		printf("res_%d=%e\n", iter+1, sc->residual);
		printf("rhores_%d=%e\n", iter+1, sqrt(rho)*bnrm2);
		printf("rhores_%d=%e\n", iter+1, rho);
		printf("check_%d=%e\n", iter+1, check);
		#endif
		if(sc->residual < sc->tolerance) {
			break;
		}
		

		/* beta      = rho(k+1) / rho(k) */
		beta = rho / rho_old;

		/* p(k+1)    = r(k+1) + beta*p(k) */
		LAUNCH(xpay)(devZ, beta, n, devP);
		printError();

	}
	hipDeviceSynchronize();
	printf("res_%d=%e\n", iter+1, sc->residual);

	// copy x back
	CHECK_CUDA_ERROR(hipMemcpy(x, devX, fvecSize, hipMemcpyDeviceToHost ));

	/* Store the number of iterations and the 
	 * time for the sparse matrix vector
	 * product which is the most expensive 
	 * function in the whole CG algorithm. */
	sc->iter = iter;
	sc->timeMatvec = timeMatvec;

	/* Clean up */
	hipFree(devR);
	hipFree(devP);
	hipFree(devQ);
	hipFree(devB);
	hipFree(devX);
	hipFree(devZ);
	hipFree(devDiag);
	hipFree(devIndices);
	hipFree(devData);
	hipFree(devLength);
}
